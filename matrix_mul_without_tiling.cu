/**
*Developed By Karan Bhagat
*March 2017
**/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//cuda kernel for multiplying two matrices without tiling
__global__ void matrix_mul_kernel(int* a, int* b, int* c, int a_rows, int a_columns, int b_columns)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	//check if thread directly maps to the dimensions of resulting matrix
	if (row < a_rows && col < b_columns)
	{
		int result = 0;
		int k;
		for (k = 0; k < a_columns; k++)
		{
			result += (a[row * a_columns + k] * b[k * b_columns + col]);
		}
		c[row * b_columns + col] = result;
	}
}

void build_matrix(FILE *file, int* mat, int rows, int columns);

int main(int argc, char **argv)
{
	//check for filenames and matrices' dimensions
	if (argc != 6)
	{
		printf("Usage : ./matrix_mul_tiling <fileA> <fileB> <A_rows> <A_columns> <B_columns>");
		exit(1);
	}
	
	char* fileA_name = argv[1];//matrix A filename
	char* fileB_name = argv[2];//matrix B filename

	// a_columns can also be perceived as b_rows
	int a_rows, a_columns, b_columns;
	
	//read matrix A and B's dimensions
	sscanf(argv[3], "%d", &a_rows);
	sscanf(argv[4], "%d", &a_columns);
	sscanf(argv[5], "%d", &b_columns);
	
	FILE *fileA = fopen(fileA_name, "r");
	FILE *fileB = fopen(fileB_name, "r");

	//declare host and device matrices pointers
	int* mat_a;
	int* mat_b;
	int* mat_c;
	int* d_mat_a;
	int* d_mat_b;
	int* d_mat_c;
	
	//allocate memory for host matrices
	mat_a = (int*)malloc(a_rows * a_columns * sizeof(int));
	mat_b = (int*)malloc(a_columns * b_columns * sizeof(int));
	mat_c = (int*)malloc(a_rows * b_columns * sizeof(int));
	
	int i, j;
	
	build_matrix(fileA, mat_a, a_rows, a_columns);
	build_matrix(fileB, mat_b, a_columns, b_columns);
	
	//declare dimensions for the grid and block
	dim3 dimBlock(2,2);
	dim3 dimGrid((int)ceil(b_columns/2),(int)ceil(a_rows/2));
	
	const size_t size_a = a_rows * a_columns * sizeof(int);
	const size_t size_b = a_columns * b_columns * sizeof(int);
	const size_t size_c = a_rows * b_columns * sizeof(int);

	//allocate matrices memeory on device
	hipMalloc((void **)&d_mat_a, size_a);
	hipMalloc((void **)&d_mat_b, size_b);
	hipMalloc((void **)&d_mat_c, size_c);

	//copy A and B matrices from host to device
	hipMemcpy(d_mat_a, mat_a, size_a, hipMemcpyHostToDevice);
	hipMemcpy(d_mat_b, mat_b, size_b, hipMemcpyHostToDevice);

	//execute cuda kernel
	matrix_mul_kernel<<<dimGrid, dimBlock>>>(d_mat_a, d_mat_b, d_mat_c, a_rows, a_columns, b_columns);

	//copy the compute matrix C from device to host
	hipMemcpy(mat_c, d_mat_c, size_c, hipMemcpyDeviceToHost);
	
	//free cuda memory
	hipFree(d_mat_a);
	hipFree(d_mat_b);
	hipFree(d_mat_c);

	//print the resulting matrix
	for (i = 0; i < a_rows; i++)
	{
		for (j = 0; j < b_columns; j++)
		{
			printf("%d ", mat_c[i * b_columns + j]);
		}
		printf("\n");
	}
}

//build matrix from the file
void build_matrix(FILE *file, int* mat, int rows, int columns)
{
	int i, j;
	for (i = 0; i < rows; i++)
	{
		for (j = 0; j < columns; j++) 
		{
			fscanf(file, "%d", &mat[i * columns + j]);
		}
	}
}